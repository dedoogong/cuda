/*
Two Dimensional (2D) Image Convolution : A Basic Approch
Image Convolution is a very basic operation in the field of Image Processing. 
It is required in many algorithms in Image Processing. Also it is very compute intensive task as 
it involves operation with pixels.
Its a transformation which involves a Mask and an Image on which that mask will be performing the operation.


General steps of Convolution:
The center point of Mask is first placed on to the Image Pixel.
Each pixel of Mask is multiplied to corresponding pixel of Image.
A complete sum (Cumulative sum) of all multiplications performed between Mask and Image pixels are then 
put in the related Image pixel value as a result of Convolution.
Following is the sample code of Image convolution. The Mask_width, Mask_height are set to 3 as 
its a 3 X 3 2D array with all values set to 1. Also width and height are also set to 3 
as I considered image of size 3 X 3 only for the sake of the example. 
You may change the values as per your need.
*/



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define Mask_width 3
#define Mask_height 3
#define width 3
#define height 3
float convolution_2D_OnHost(float * N,float * M,int i,int j);

int main()
{
 float * input;
 float * Mask;
 float * output;

 input=(float *)malloc(sizeof(float)*width*height);
 Mask=(float *)malloc(sizeof(float)*Mask_width*Mask_height);
 output=(float *)malloc(sizeof(float)*width*height);

 for(int i=0;i<width*height;i++)
 {
  input[i]=1.0;
 }
 for(int i=0;i<Mask_width*Mask_height;i++)
 {
  Mask[i]=1.0;
 }

 printf("\nInput Array:\n");
 for(int i=0;i<width*height;i++)
 {
  if(i>0 && (i%width==0))
  printf("\n");
  printf("%0.2f ",input[i]);

 }printf("\n");

 printf("\nMask:\n");
  for(int i=0;i<Mask_width*Mask_height;i++)
  {
   if(i>0 && (i%Mask_width==0))
   printf("\n");
   printf("%0.2f ",Mask[i]);

  }printf("\n");

  for(int i=0;i<width;i++)
  {
   for(int j=0;j<height;j++)
   {
    output[(i*width)+j]=convolution_2D_OnHost(input,Mask,i,j);
   }
  }

  printf("\nOutput:\n");
  for(int i=0;i<width*height;i++)
    {
    if(i>0 && (i%width==0))
     printf("\n");
    printf("%d = %0.2f \t",i,*(output+i));
    }

free(input);
free(Mask);
free(output);
return 0;
}

float convolution_2D_OnHost(float * N,float * M,int i,int j)
{
 float Pvalue=0.0;
 int N_start_point_i=i-(Mask_width/2);
 int N_start_point_j=j-(Mask_height/2);

 for(int k=0;k<Mask_width;k++)
 {
  for(int l=0;l<Mask_height;l++)
  {
   if(((N_start_point_i+k)>=0) && ((N_start_point_i+k)<width)&&((N_start_point_j+l)>=0)&&((N_start_point_j+l)<height))
   {
   Pvalue+=N[(N_start_point_i+k)*width+(N_start_point_j+l)]*M[(k*Mask_width)+l];
   }
  }
 }
 return(Pvalue);
}