/*
Two Dimensional (2D) Image Convolution in CUDA by Shared & Constant Memory: An Optimized way
After learning the concept of two dimension (2D) Convolution and its implementation in C language;
the next step is to learn to optimize it. As Convolution is one of the most Compute Intensive task in 
Image Processing, it is always better to save time required for it. So, today I am going to share a technique 
to optimize the Convolution process by using CUDA. Here we will use Shared Memory and Constant Memory resources
available in CUDA to get fastest implementation of Convolution.

Here is the strategy, 
We will store convolution mask in Constant Memory as all threads will need the same Mask. Constant Memory is 
the off-chip memory present on GPU card which is read-only memory and faster than Global memory. The Mask is 
passed to kernel with the keyword "__restrict__". Check for it in the kernel definition.
Instead of accessing complete image and performing the convolution operation on it; we will fix TILE size. 
The image pixels are copied to this TILE first and then convolution is performed accordingly. 
This TILE will be stored in the Shared Memory of GPU card which is on-chip memory and faster than 
the Global Memory.
Following is the sample code of the implementation. You can also get it from my GitHub repository.

*/
/*
 * Title: 2D Image Convolution on GPU by using Shared Memory and Constant Memory.
 *
 * Image Size: 2048 X 2048
 * Mask Size: 64 X 64
 * TILE_WIDTH 32
 *
 *
 * */
#include<stdio.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<assert.h>

#define wbCheck(stmt)  do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)


#define Mask_width  64
#define Mask_height 64
#define Mask_radius_x Mask_width/2
#define Mask_radius_y Mask_height/2
#define TILE_WIDTH 32  //16 X 16 TILE
#define w_x (TILE_WIDTH + Mask_width - 1) //Shared Memory Elements needed to load as per Mask Size
#define w_y (TILE_WIDTH + Mask_height - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE
__global__ void convolution(float *I, const float* __restrict__ M, float *P,int channels, int width, int height)
{
   __shared__ float N_ds[w_y][w_x];
   int k;
   for (k = 0; k < channels; k++)
   {
      //1. Phase to Load Data into Shared Memory. Each Thread loads multiple elements indexed by each Batch loading
    //1.dest: RMO ID 2. destY & destX: Row and Column of Shared Memory
    //3. srcY & srcX: Indexes to fetch data from input Image
    //4. src: RMO index of Input Image

    // First batch loading
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
         destY = dest / w_x, destX = dest % w_x,
         srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius_x,
         srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius_y,
         src = (srcY * width + srcX) * channels + k;
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_ds[destY][destX] = I[src];
      else
         N_ds[destY][destX] = 0.0;

        for (int iter=1; iter <= (w_x * w_y) / (TILE_WIDTH*TILE_WIDTH); iter++)
        {
          // Second batch loading
          dest = threadIdx.y * TILE_WIDTH + threadIdx.x + iter*(TILE_WIDTH * TILE_WIDTH);
            destY = dest / w_x, destX = dest % w_x;
            srcY  = blockIdx.y * TILE_WIDTH + destY - Mask_radius_x;
            srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius_y;
            src = (srcY * width + srcX) * channels + k;
            if (destY < w_y && destX < w_x)
            {
                if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                    N_ds[destY][destX] = I[src];
                else
                    N_ds[destY][destX] = 0.0;
            }
        }
      __syncthreads();

      float accum = 0;
      int y, x;
      for (y = 0; y < Mask_width; y++)
         for (x = 0; x < Mask_width; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
      y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         //P[(y * width + x) * channels + k] = clamp(accum);
       P[(y * width + x) * channels + k] = accum;
      __syncthreads();
   }
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j);

int main() {

    int maskRows=Mask_height; // Set it as per requirement of 64 X 32
    int maskColumns=Mask_width;

    int imageChannels=1;
    int imageWidth=2048;
    int imageHeight=2048;

    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;
    float * outputImageOnHost;

    hostMaskData = (float *) malloc(sizeof(float)*maskRows*maskColumns);
    for(int i=0;i<maskRows*maskColumns;i++)//To set Mask of size 5*5 which has all values as 1
    {
     hostMaskData[i]=1.0;
    }

//Comment this assert code for 64 X 32
    //assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    //assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    //To store Memory
    hostInputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    outputImageOnHost = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    for(int i=0;i<imageWidth*imageHeight;i++)//To set Image data as 1.0
    {
     hostInputImageData[i]=1.0;
    }

    wbCheck(hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
    wbCheck(hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
    wbCheck(hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float)));

    wbCheck(hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice));

    dim3 dimGrid(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
                                       imageChannels, imageWidth, imageHeight);

 hipError_t err1 = hipPeekAtLastError();
 hipDeviceSynchronize();
 printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);

    //printf("\n Output from Device:\n");
#if 0  //comment this to run the portion of code
    for(int i=0;i<imageWidth*imageHeight;i++)
    {
        if(i>0 && (i%imageWidth==0))
            printf("\n");
      printf("%0.2f \t",*(hostOutputImageData+i));

      }
#endif

    //Convolution on Host
    for(int i=0;i<imageWidth;i++)
          {
           for(int j=0;j<imageHeight;j++)
           {
            outputImageOnHost[(i*imageWidth)+j]=convolution_2D_OnHost(hostInputImageData,hostMaskData,imageWidth,imageHeight,i,j);
           }
          }

    //printf("\n Output from Host:\n");
#if 0  //comment this to run the portion of code
    for(int i=0;i<imageWidth*imageHeight;i++)
      {
      if(i>0 && (i%imageWidth==0))
       printf("\n");
      printf("%0.2f \t",*(outputImageOnHost+i));

      }
#endif


        for(int i=0;i<imageWidth*imageHeight;i++)
        {
         if(outputImageOnHost[i]!=hostOutputImageData[i])
         {
           printf("\nMismatch at (Row,Col) = [%d][%d], hostComputed[]: %0.0f And device[]: %0.0f", i / imageWidth, i % imageHeight, outputImageOnHost[i], hostOutputImageData[i]);
            }
        }

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    free(hostInputImageData);
    free(hostOutputImageData);

    printf("\n Number of Threads Per Block created in code: %d",TILE_WIDTH*TILE_WIDTH);
    printf("\n Number of Blocks Created :%d",(((imageWidth-1)/TILE_WIDTH)+1)*(((imageWidth-1)/TILE_WIDTH)+1));
    printf("No Error");
    return 0;
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j)
{
 float Pvalue=0.0;
 int N_start_point_i = i - (Mask_width/2);
 int N_start_point_j = j - (Mask_height/2);

 for(int k=0;k<Mask_width;k++)
 {
  for(int l=0;l<Mask_height;l++)
  {
   if(((N_start_point_i+k)>=0) && ((N_start_point_i+k)<width)&&((N_start_point_j+l)>=0)&&((N_start_point_j+l)<height))
   {
       Pvalue+=N[(N_start_point_i+k)*width+(N_start_point_j+l)]*M[(k*Mask_width)+l];
   }
  }
 }

// return(clamp(Pvalue));
 return((Pvalue));
}
