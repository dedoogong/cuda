/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

float fx = 1.0f, fy = 1.0f, fz = 1.0f;
const int mx = 64, my = 64, mz = 64;
  
// shared memory tiles will be m*-by-*Pencils
// sPencils is used when each thread calculates the derivative at one point
// lPencils is used for coalescing in y and z where each thread has to 
//     calculate the derivative at mutiple points
const int sPencils = 4;  // small # pencils
const int lPencils = 32; // large # pencils
  
dim3 grid[3][2], block[3][2];

// stencil coefficients
__constant__ float c_ax, c_bx, c_cx, c_dx;
__constant__ float c_ay, c_by, c_cy, c_dy;
__constant__ float c_az, c_bz, c_cz, c_dz;
 
// host routine to set constant data
void setDerivativeParameters()
{
  // check to make sure dimensions are integral multiples of sPencils
  if ((mx % sPencils != 0) || (my %sPencils != 0) || (mz % sPencils != 0)) {
    printf("'mx', 'my', and 'mz' must be integral multiples of sPencils\n");
    exit(1);
  }
  
  if ((mx % lPencils != 0) || (my % lPencils != 0)) {
    printf("'mx' and 'my' must be multiples of lPencils\n");
    exit(1);
  }

  // stencil weights (for unit length problem)
  float dsinv = mx-1.f;
  
  float ax =  4.f / 5.f   * dsinv;
  float bx = -1.f / 5.f   * dsinv;
  float cx =  4.f / 105.f * dsinv;
  float dx = -1.f / 280.f * dsinv;
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_ax), &ax, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_bx), &bx, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_cx), &cx, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_dx), &dx, sizeof(float), 0, hipMemcpyHostToDevice) );

  dsinv = my-1.f;
  
  float ay =  4.f / 5.f   * dsinv;
  float by = -1.f / 5.f   * dsinv;
  float cy =  4.f / 105.f * dsinv;
  float dy = -1.f / 280.f * dsinv;
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_ay), &ay, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_by), &by, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_cy), &cy, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_dy), &dy, sizeof(float), 0, hipMemcpyHostToDevice) );

  dsinv = mz-1.f;
  
  float az =  4.f / 5.f   * dsinv;
  float bz = -1.f / 5.f   * dsinv;
  float cz =  4.f / 105.f * dsinv;
  float dz = -1.f / 280.f * dsinv;
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_az), &az, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_bz), &bz, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_cz), &cz, sizeof(float), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_dz), &dz, sizeof(float), 0, hipMemcpyHostToDevice) );

  // Execution configurations for small and large pencil tiles

  grid[0][0]  = dim3(my / sPencils, mz, 1);
  block[0][0] = dim3(mx, sPencils, 1);

  grid[0][1]  = dim3(my / lPencils, mz, 1);
  block[0][1] = dim3(mx, sPencils, 1);

  grid[1][0]  = dim3(mx / sPencils, mz, 1);
  block[1][0] = dim3(sPencils, my, 1);

  grid[1][1]  = dim3(mx / lPencils, mz, 1);
  // we want to use the same number of threads as above,
  // so when we use lPencils instead of sPencils in one
  // dimension, we multiply the other by sPencils/lPencils
  block[1][1] = dim3(lPencils, my * sPencils / lPencils, 1);

  grid[2][0]  = dim3(mx / sPencils, my, 1);
  block[2][0] = dim3(sPencils, mz, 1);

  grid[2][1]  = dim3(mx / lPencils, my, 1);
  block[2][1] = dim3(lPencils, mz * sPencils / lPencils, 1);
}

void initInput(float *f, int dim)
{
  const float twopi = 8.f * (float)atan(1.0);

  for (int k = 0; k < mz; k++) {
    for (int j = 0; j < my; j++) {
      for (int i = 0; i < mx; i++) {
        switch (dim) {
          case 0: 
            f[k*mx*my+j*mx+i] = cos(fx*twopi*(i-1.f)/(mx-1.f));
            break;
          case 1:
            f[k*mx*my+j*mx+i] = cos(fy*twopi*(j-1.f)/(my-1.f));
            break;
          case 2:
            f[k*mx*my+j*mx+i] = cos(fz*twopi*(k-1.f)/(mz-1.f));
            break;
        }
      }
    }
  }     
}

void initSol(float *sol, int dim)
{
  const float twopi = 8.f * (float)atan(1.0);

  for (int k = 0; k < mz; k++) {
    for (int j = 0; j < my; j++) {
      for (int i = 0; i < mx; i++) {
        switch (dim) {
          case 0: 
            sol[k*mx*my+j*mx+i] = -fx*twopi*sin(fx*twopi*(i-1.f)/(mx-1.f));
            break;
          case 1:
            sol[k*mx*my+j*mx+i] = -fy*twopi*sin(fy*twopi*(j-1.f)/(my-1.f));
            break;
          case 2:
            sol[k*mx*my+j*mx+i] = -fz*twopi*sin(fz*twopi*(k-1.f)/(mz-1.f));
            break;
        }
      }
    }
  }    
}

void checkResults(double &error, double &maxError, float *sol, float *df)
{
  // error = sqrt(sum((sol-df)**2)/(mx*my*mz))
  // maxError = maxval(abs(sol-df))
  maxError = 0;
  error = 0;
  for (int k = 0; k < mz; k++) {
    for (int j = 0; j < my; j++) {
      for (int i = 0; i < mx; i++) {
        float s = sol[k*mx*my+j*mx+i];
        float f = df[k*mx*my+j*mx+i];
        //printf("%d %d %d: %f %f\n", i, j, k, s, f);
        error += (s-f)*(s-f);
        if (fabs(s-f) > maxError) maxError = fabs(s-f);
      }
    }
  }
  error = sqrt(error / (mx*my*mz));
}
  

// -------------
// x derivatives
// -------------

__global__ void derivative_x(float *f, float *df)
{  
  __shared__ float s_f[sPencils][mx+8]; // 4-wide halo

  int i   = threadIdx.x;
  int j   = blockIdx.x*blockDim.y + threadIdx.y;
  int k  = blockIdx.y;
  int si = i + 4;       // local i for shared memory access + halo offset
  int sj = threadIdx.y; // local j for shared memory access

  int globalIdx = k * mx * my + j * mx + i;

  s_f[sj][si] = f[globalIdx];

  __syncthreads();

  // fill in periodic images in shared memory array 
  if (i < 4) {
    s_f[sj][si-4]  = s_f[sj][si+mx-5];
    s_f[sj][si+mx] = s_f[sj][si+1];   
  }

  __syncthreads();
  
  df[globalIdx] = 
    ( c_ax * ( s_f[sj][si+1] - s_f[sj][si-1] )
    + c_bx * ( s_f[sj][si+2] - s_f[sj][si-2] )
    + c_cx * ( s_f[sj][si+3] - s_f[sj][si-3] )
    + c_dx * ( s_f[sj][si+4] - s_f[sj][si-4] ) );
}


// this version uses a 64x32 shared memory tile, 
// still with 64*sPencils threads

__global__ void derivative_x_lPencils(float *f, float *df)
{
  __shared__ float s_f[lPencils][mx+8]; // 4-wide halo
  
  int i     = threadIdx.x;
  int jBase = blockIdx.x*lPencils;
  int k     = blockIdx.y;
  int si    = i + 4; // local i for shared memory access + halo offset

  for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
    int globalIdx = k * mx * my + (jBase + sj) * mx + i;      
    s_f[sj][si] = f[globalIdx];
  }

  __syncthreads();

  // fill in periodic images in shared memory array 
  if (i < 4) {
    for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
      s_f[sj][si-4]  = s_f[sj][si+mx-5];
      s_f[sj][si+mx] = s_f[sj][si+1];
    }
  }

  __syncthreads();

  for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
     int globalIdx = k * mx * my + (jBase + sj) * mx + i;      
     df[globalIdx] = 
      ( c_ax * ( s_f[sj][si+1] - s_f[sj][si-1] )
      + c_bx * ( s_f[sj][si+2] - s_f[sj][si-2] )
      + c_cx * ( s_f[sj][si+3] - s_f[sj][si-3] )
      + c_dx * ( s_f[sj][si+4] - s_f[sj][si-4] ) );
  }
}

// -------------
// y derivatives
// -------------

__global__ void derivative_y(float *f, float *df)
{
  __shared__ float s_f[my+8][sPencils];

  int i  = blockIdx.x*blockDim.x + threadIdx.x;
  int j  = threadIdx.y;
  int k  = blockIdx.y;
  int si = threadIdx.x;
  int sj = j + 4;

  int globalIdx = k * mx * my + j * mx + i;

  s_f[sj][si] = f[globalIdx];
  
  __syncthreads();

  if (j < 4) {
    s_f[sj-4][si]  = s_f[sj+my-5][si];
    s_f[sj+my][si] = s_f[sj+1][si];
  }

  __syncthreads();

  df[globalIdx] = 
    ( c_ay * ( s_f[sj+1][si] - s_f[sj-1][si] )
    + c_by * ( s_f[sj+2][si] - s_f[sj-2][si] )
    + c_cy * ( s_f[sj+3][si] - s_f[sj-3][si] )
    + c_dy * ( s_f[sj+4][si] - s_f[sj-4][si] ) );
}

// y derivative using a tile of 32x64,
// launch with thread block of 32x8
__global__ void derivative_y_lPencils(float *f, float *df)
{
  __shared__ float s_f[my+8][lPencils];

  int i  = blockIdx.x*blockDim.x + threadIdx.x;
  int k  = blockIdx.y;
  int si = threadIdx.x;
  
  for (int j = threadIdx.y; j < my; j += blockDim.y) {
    int globalIdx = k * mx * my + j * mx + i;
    int sj = j + 4;
    s_f[sj][si] = f[globalIdx];
  }

  __syncthreads();

  int sj = threadIdx.y + 4;
  if (sj < 8) {
     s_f[sj-4][si]  = s_f[sj+my-5][si];
     s_f[sj+my][si] = s_f[sj+1][si];   
  }

  __syncthreads();

  for (int j = threadIdx.y; j < my; j += blockDim.y) {
    int globalIdx = k * mx * my + j * mx + i;
    int sj = j + 4;
    df[globalIdx] = 
      ( c_ay * ( s_f[sj+1][si] - s_f[sj-1][si] )
      + c_by * ( s_f[sj+2][si] - s_f[sj-2][si] )
      + c_cy * ( s_f[sj+3][si] - s_f[sj-3][si] )
      + c_dy * ( s_f[sj+4][si] - s_f[sj-4][si] ) );
  }
}


// ------------
// z derivative
// ------------

__global__ void derivative_z(float *f, float *df)
{
  __shared__ float s_f[mz+8][sPencils];

  int i  = blockIdx.x*blockDim.x + threadIdx.x;
  int j  = blockIdx.y;
  int k  = threadIdx.y;
  int si = threadIdx.x;
  int sk = k + 4; // halo offset

  int globalIdx = k * mx * my + j * mx + i;

  s_f[sk][si] = f[globalIdx];

  __syncthreads();

  if (k < 4) {
     s_f[sk-4][si]  = s_f[sk+mz-5][si];
     s_f[sk+mz][si] = s_f[sk+1][si];
  }

  __syncthreads();

  df[globalIdx] = 
    ( c_az * ( s_f[sk+1][si] - s_f[sk-1][si] )
    + c_bz * ( s_f[sk+2][si] - s_f[sk-2][si] )
    + c_cz * ( s_f[sk+3][si] - s_f[sk-3][si] )
    + c_dz * ( s_f[sk+4][si] - s_f[sk-4][si] ) );
}

__global__ void derivative_z_lPencils(float *f, float *df)
{
  __shared__ float s_f[mz+8][lPencils];

  int i  = blockIdx.x*blockDim.x + threadIdx.x;
  int j  = blockIdx.y;
  int si = threadIdx.x;

  for (int k = threadIdx.y; k < mz; k += blockDim.y) {
    int globalIdx = k * mx * my + j * mx + i;
    int sk = k + 4;
    s_f[sk][si] = f[globalIdx];
  }

  __syncthreads();

  int k = threadIdx.y + 4;
  if (k < 8) {
     s_f[k-4][si]  = s_f[k+mz-5][si];
     s_f[k+mz][si] = s_f[k+1][si];
  }

  __syncthreads();

  for (int k = threadIdx.y; k < mz; k += blockDim.y) {
    int globalIdx = k * mx * my + j * mx + i;
    int sk = k + 4;
    df[globalIdx] = 
        ( c_az * ( s_f[sk+1][si] - s_f[sk-1][si] )
        + c_bz * ( s_f[sk+2][si] - s_f[sk-2][si] )
        + c_cz * ( s_f[sk+3][si] - s_f[sk-3][si] )
        + c_dz * ( s_f[sk+4][si] - s_f[sk-4][si] ) );  
  }
}

// Run the kernels for a given dimension. One for sPencils, one for lPencils
void runTest(int dimension)
{
  void (*fpDeriv[2])(float*, float*);

  switch(dimension) {
    case 0:
      fpDeriv[0] = derivative_x;
      fpDeriv[1] = derivative_x_lPencils;
      break;
    case 1:
      fpDeriv[0] = derivative_y;
      fpDeriv[1] = derivative_y_lPencils;
      break;
    case 2:
      fpDeriv[0] = derivative_z;
      fpDeriv[1] = derivative_z_lPencils;
      break;
  }

  int sharedDims[3][2][2] = { mx, sPencils, 
                              mx, lPencils,
                              sPencils, my,
                              lPencils, my,
                              sPencils, mz,
                              lPencils, mz };

  float *f = new float[mx*my*mz];
  float *df = new float[mx*my*mz];
  float *sol = new float[mx*my*mz];                           
    
  initInput(f, dimension);
  initSol(sol, dimension);

  // device arrays
  int bytes = mx*my*mz * sizeof(float);
  float *d_f, *d_df;
  checkCuda( hipMalloc((void**)&d_f, bytes) );
  checkCuda( hipMalloc((void**)&d_df, bytes) );

  const int nReps = 20;
  float milliseconds;
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  double error, maxError;

  printf("%c derivatives\n\n", (char)(0x58 + dimension));

  for (int fp = 0; fp < 2; fp++) { 
    checkCuda( hipMemcpy(d_f, f, bytes, hipMemcpyHostToDevice) );  
    checkCuda( hipMemset(d_df, 0, bytes) );
    
    fpDeriv[fp]<<<grid[dimension][fp],block[dimension][fp]>>>(d_f, d_df); // warm up
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < nReps; i++)
       fpDeriv[fp]<<<grid[dimension][fp],block[dimension][fp]>>>(d_f, d_df);
    
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&milliseconds, startEvent, stopEvent) );

    checkCuda( hipMemcpy(df, d_df, bytes, hipMemcpyDeviceToHost) );
        
    checkResults(error, maxError, sol, df);

    printf("  Using shared memory tile of %d x %d\n", 
           sharedDims[dimension][fp][0], sharedDims[dimension][fp][1]);
    printf("   RMS error: %e\n", error);
    printf("   MAX error: %e\n", maxError);
    printf("   Average time (ms): %f\n", milliseconds / nReps);
    printf("   Average Bandwidth (GB/s): %f\n\n", 
           2.f * 1e-6 * mx * my * mz * nReps * sizeof(float) / milliseconds);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );

  checkCuda( hipFree(d_f) );
  checkCuda( hipFree(d_df) );

  delete [] f;
  delete [] df;
  delete [] sol;
}


// This the main host code for the finite difference 
// example.  The kernels are contained in the derivative_m module

int main(void)
{
  // Print device and precision
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, 0) );
  printf("\nDevice Name: %s\n", prop.name);
  printf("Compute Capability: %d.%d\n\n", prop.major, prop.minor);

  setDerivativeParameters(); // initialize 

  runTest(0); // x derivative
  runTest(1); // y derivative
  runTest(2); // z derivative

  return 0;
}
