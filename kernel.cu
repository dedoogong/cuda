#include "hip/hip_runtime.h"
//build commnad : nvcc -std=c++11 -lcudnn -lcublas kernel.cu -o kernel

#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include "common/common.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

///////////////////////////////////////////////////////////////////////

__global__ void staticReverse(int *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int trail = n-t-1;
  s[t] = d[t];// write step : global to shared
  __syncthreads();
  d[t] = s[trail]; // read step : shared to global
}

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int shared_memory_reverse(void)
{
  const int n = 64;
  int h_a[n], h_r[n], h_d[n];

  for (int i = 0; i < n; i++) {
    h_a[i] = i;
    h_r[i] = n-i-1;
    h_d[i] = 0;
  }
  printf("original array elemtns\n");

  for (int i = 0; i < n; i++) {
      printf("%d ",a[i]);
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));

  // ���� ���� �޸� ����
  hipMemcpy(d_d, h_a, n*sizeof(int), hipMemcpyHostToDevice);
  staticReverse<<<1,n>>>(d_d, n);
  hipMemcpy(h_d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);

  // ���� ���� �޸� ����
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

  printf("\nreverse results\n");
  int flag=1;
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]){ flag=0; printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);}
    else printf("%d ",r[i]);
  if(flag)printf("\nall array elements are correctly reversed\n");

}


///////////////////////////////////////////////////////////////////////

__global__ void kernel(float *a, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float s = sinf(x);
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s*s+c*c);
}

float maxError(float *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}


int overlap(int argc, char **argv)
{
  const int blockSize = 256, nStreams = 4;// blockSize=threadCount
  const int n = 4 * 1024 * blockSize * nStreams;
  const int streamSize = n / nStreams;// == one stream size == 4 * 1024 * blockSize
  const int streamBytes = streamSize * sizeof(float);
  const int total_bytes = n * sizeof(float);

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // ȣ��Ʈ  ���� �޸𸮿� ����̽� �޸� �Ҵ�
  float *a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, total_bytes) );      // host pinned
  checkCuda( hipMalloc((void**)&d_a, total_bytes) ); // device

  float ms; // milliseconds Ÿ�̸�

  // �̺�Ʈ �� ��Ʈ�� ����
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // �⺻ ���̽� - ������ �޸� ���۰� Ŀ�� ȣ��
  memset(a, 0, total_bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpy(d_a, a, total_bytes, hipMemcpyHostToDevice) );
  kernel<<<n/blockSize, blockSize>>>(d_a, 0);//gridSize=4*1024(blockCount)
  checkCuda( hipMemcpy(a, d_a, total_bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // �񵿱� ���� 1: [����-Ŀ��ȣ��-����]�� ������ �ݺ� ����
  memset(a, 0, total_bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // �񵿱� ���� 2:
  // ���� ����, Ŀ�� ȣ�� ����, ���� ������ ������ ����
  memset(a, 0, total_bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // �޸� ����
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}

///////////////////////////////////////////////////////////////////////
void profileCopies(float        *h_a,
                   float        *h_b,
                   float        *d,
                   unsigned int  n,
                   char         *desc)
{
  printf("\n%s transfers\n", desc);

  unsigned int bytes = n * sizeof(float);

  hipEvent_t startEvent, stopEvent;

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  float time;
  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  for (int i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      printf("*** %s transfers failed ***", desc);
      break;
    }
  }

  // �̺�Ʈ ����
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
}

int data_transfer_pageable_vs_pinned()
{
  unsigned int nElements = 4*1024*1024;
  const unsigned int bytes = nElements * sizeof(float);

  //ȣ��Ʈ �迭
  float *h_aPageable, *h_bPageable;
  float *h_aPinned, *h_bPinned;

  //����� ��  �迭
  float *d_a;

  //�Ҵ� �� �ʱ�ȭ
  h_aPageable = (float*)malloc(bytes);                    // ȣ��Ʈ pageable �޸� �Ҵ�
  h_bPageable = (float*)malloc(bytes);                    // ȣ��Ʈ pageable �޸� �Ҵ�
  checkCuda( hipHostMalloc((void**)&h_aPinned, bytes) ); // ȣ��Ʈ pinned �޸� �Ҵ�
  checkCuda( hipHostMalloc((void**)&h_bPinned, bytes) ); // ȣ��Ʈ pinned �޸� �Ҵ�
  checkCuda( hipMalloc((void**)&d_a, bytes) );           // ����̽� �޸� �Ҵ�

  for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;
  memcpy(h_aPinned, h_aPageable, bytes);
  memset(h_bPageable, 0, bytes);
  memset(h_bPinned, 0, bytes);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, 0) );

  printf("\nDevice: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));

  // ���� �޸� ���� ���� ��
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");

  printf("\n");

  // �޸� ����
  hipFree(d_a);
  hipHostFree(h_aPinned);
  hipHostFree(h_bPinned);
  free(h_aPageable);
  free(h_bPageable);

  return 0;
}


///////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void offset(T* a, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  int blockSize = 256;
  float ms;

  T *d_a;
  hipEvent_t startEvent, stopEvent;

  int n = nMB*1024*1024/sizeof(T);

  // NB:  d_a(33*nMB)
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Offset, Bandwidth (GB/s):\n");

  offset<<<n/blockSize, blockSize>>>(d_a, 0); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    offset<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  printf("\n");
  printf("Stride, Bandwidth (GB/s):\n");

  stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
  for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}
//------------------------------------------------------------------
int coaleascing(int argc, char **argv)
{
  int nMB = 4;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }

  hipDeviceProp_t prop;

  checkCuda( hipSetDevice(deviceId) );
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);

  printf("%s Precision\n", bFp64 ? "Double" : "Single");

  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);
}

///////////////////////////////////////////////////////////////////////
const int N = 1 << 20;

__global__ void kernel_target(float *x, int n){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}
void *launch_kernel(void *dummy){
    float *data;
    hipMalloc(&data, N * sizeof(float));
    kernel_target<<<1, 64>>>(data, N);
    hipStreamSynchronize(0);
    return NULL;
}
int multithread(){
    const int num_threads = 8;
    pthread_t threads[num_threads];
    for (int i = 0; i < num_threads; i++) {
        if (pthread_create(&threads[i], NULL, launch_kernel, 0)) {
            fprintf(stderr, "Error creating threadn");
     }
   }
    for (int i = 0; i < num_threads; i++) {
        if(pthread_join(threads[i], NULL)) {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }
    hipDeviceReset();
    return 0;
}

///////////////////////////////////////////////////////////////////////

int ROWS = 1024;
int COLS = 1024;

void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rand_max = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    for (j = 0; j < N; j++){//��
        for (i = 0; i < M; i++){//��
            double drand = (double)rand();
            A[j * M + i] = (drand / rand_max) * 100.0; //0-100 ���� ��
        }
    }
    *outA = A;
}

int cublasMM(int argc, char **argv)
{
    int i, j;
    float *A, *dA;
    float *B, *dB;
    float *C, *dC;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    alpha = 3.0f;
    beta = 4.0f;
    int N = ROWS;
    int M = COLS;
    // �Է� ������ �ʱ�ȭ
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_dense_matrix(N, M, &B);
    C = (float *)malloc(sizeof(float) * M * M);
    memset(C, 0x00, sizeof(float) * M * M);

    // cuBLAS �ڵ鷯 ����
    CHECK_CUBLAS(hipblasCreate(&handle));

    // ����̽� �޸� �Ҵ�
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(float) * M * M));

    // ����̽��� ������ ����
    CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M));
    CHECK_CUBLAS(hipblasSetMatrix(N, M, sizeof(float), B, N, dB, N));
    CHECK_CUBLAS(hipblasSetMatrix(M, M, sizeof(float), C, M, dC, M));

    // ���-���� �� ����
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, N, &alpha,
                dA, M, dB, N, &beta, dC, M));

    // ��� �� ��ȯ �� Ȯ��
    CHECK_CUBLAS(hipblasGetMatrix(M, M, sizeof(float), dC, M, C, M));

    for (j = 0; j < 10; j++)
    {
        for (i = 0; i < 10; i++)
        {
            printf("%2.2f ", C[j * M + i]);
        }
        printf("...\n");
    }

    printf("...\n");

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}

///////////////////////////////////////////////////////////////////////

int cublasMMAsync(int argc, char **argv)
{
    int i, j;
    float *A, *dA;
    float *B, *dB;
    float *C, *dC;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;
    hipStream_t stream = 0;

    alpha = 3.0f;
    beta = 4.0f;
    int N = ROWS;
    int M = COLS;
    // �Է� ������ �ʱ�ȭ
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_dense_matrix(N, M, &B);
    C = (float *)malloc(sizeof(float) * M * M);
    memset(C, 0x00, sizeof(float) * M * M);

    // cuBLAS �ڵ鷯 ����
    CHECK_CUBLAS(hipblasCreate(&handle));
    CHECK(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));

    // ����̽� �޸� �Ҵ�
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(float) * M * M));

    // ����̽��� ������ �񵿱� ����
    CHECK_CUBLAS(hipblasSetMatrixAsync(M, N, sizeof(float), A, M, dA, M, stream));
    CHECK_CUBLAS(hipblasSetMatrixAsync(N, M, sizeof(float), B, N, dB, N, stream));
    CHECK_CUBLAS(hipblasSetMatrixAsync(M, M, sizeof(float), C, M, dC, M, stream));

    // ���-���� �� ����
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, N, &alpha,
                dA, M, dB, N, &beta, dC, M));

    // ��� �� ��ȯ �� Ȯ��
    CHECK_CUBLAS(hipblasGetMatrixAsync(M, M, sizeof(float), dC, M, C, M,
                stream));
    CHECK(hipStreamSynchronize(stream));

    for (j = 0; j < 10; j++)
    {
        for (i = 0; i < 10; i++)
        {
            printf("%2.2f ", C[j * M + i]);
        }
        printf("...\n");
    }

    printf("...\n");

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipStreamDestroy(stream));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}

///////////////////////////////////////////////////////////////////////

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

int checkResult(float *data, const int n, const float x)
{
    for (int i = 0; i < n; i++)
    {
        if (data[i] != x)
        {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

int async(int argc, char *argv[])
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CHECK(hipGetDeviceProperties(&deviceProps, devID));
    printf("> %s running on", argv[0]);
    printf(" CUDA device [%s]\n", deviceProps.name);

    int num = 1 << 24;
    int nbytes = num * sizeof(int);
    float value = 10.0f;

    // ȣ��Ʈ �޸� �Ҵ�
    float *h_a = 0;
    CHECK(hipHostMalloc((void **)&h_a, nbytes));
    memset(h_a, 0, nbytes);

    // ����̽� �޸� �Ҵ�
    float *d_a = 0;
    CHECK(hipMalloc((void **)&d_a, nbytes));
    CHECK(hipMemset(d_a, 255, nbytes));

    // ������ ���̾ƿ� ����
    dim3 block = dim3(512);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    // �̺�Ʈ �ڵ鷯 ����
    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));

    // �񵿱� �޸� ���� �� Ŀ�� ȣ��(��� ��Ʈ�� 0����)
    CHECK(hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    kernel<<<grid, block>>>(d_a, value);
    CHECK(hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop));

    // GPU �۾��� ����Ǵ� ���� CPU�� �۾� ����
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    printf("CPU executed %lu iterations while waiting for GPU to finish\n",
           counter);

    bool bFinalResults = (bool) checkResult(h_a, num, value);

    CHECK(hipEventDestroy(stop));
    CHECK(hipHostFree(h_a));
    CHECK(hipFree(d_a));

    CHECK(hipDeviceReset());

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}


///////////////////////////////////////////////////////////////////////

#define BDIMX 32
#define BDIMY 32
#define IPAD  1

void printData(char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__global__ void setRowReadRow (int *out)
{
    // ���� ���� �޸�
    __shared__ int tile[BDIMY][BDIMX]; // x, y

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx; // x, y

    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x] ;// x, y
}

__global__ void setColReadCol (int *out)
{
    // ���� ���� �޸�
    __shared__ int tile[BDIMX][BDIMY]; // y, x

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.x][threadIdx.y] = idx;// y, x

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];// y, x
}

__global__ void setRowReadCol(int *out)
{
    // ���� ���� �޸�
    __shared__ int tile[BDIMY][BDIMX];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadColDyn(int *out)
{
    // ���� ���� �޸�
    extern  __shared__ int tile[];

    unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

    tile[row_idx] = row_idx;

    __syncthreads();

    out[row_idx] = tile[col_idx];
}

__global__ void setRowReadColPad(int *out)
{
    // ���� ���� �޸� �е�
    __shared__ int tile[BDIMY][BDIMX + IPAD];

    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadColDynPad(int *out)
{
    // ���� ���� �޸� �е�
    extern  __shared__ int tile[];

    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int col_idx = threadIdx.x * (blockDim.x + IPAD) + threadIdx.y;

    unsigned int g_idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[row_idx] = g_idx;
    __syncthreads();
    out[g_idx] = tile[col_idx];
}


int smemSquare(int argc, char **argv)
{
    // ����̽� ����
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CHECK(hipDeviceGetSharedMemConfig ( &pConfig ));
    printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");

    // �迭 ũ�� ����(2048)
    int nx = BDIMX;
    int ny = BDIMY;

    bool iprintf = 0;

    if (argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    // ���� ���� ����
    dim3 block (BDIMX, BDIMY);
    dim3 grid  (1, 1);
    printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
           block.y);

    // ����̽� �޸� �Ҵ�
    int *d_C;
    CHECK(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);

    CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set col read col   ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read row   ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col   ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMX*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col dyn", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col pad", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDynPad<<<grid, block, (BDIMX + IPAD)*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprintf)  printData("set row read col DP ", gpuRef, nx * ny);

    CHECK(hipFree(d_C));
    free(gpuRef);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////
#define DIM 128



extern __shared__ int dsmem[];

int recursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;

    for (int i = 0; i < stride; i++)
        data[i] += data[i + stride];

    return recursiveReduce(data, stride);
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4;
    }

    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    int tmpSum = 0;

    // boundary check
    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)  smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)   smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4
    int tmpSum = 0;

    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceNeighboredGmem(int *g_idata, int *g_odata,
                                     unsigned int  n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata,
                                     unsigned int  n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            smem[tid] += smem[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

int reduceSum(int argc, char **argv)
{
    // ����̽� ����
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // �ʱ�ȭ
    int size = 1 << 22;
    printf("    with array size %d  ", size);

    // ���� ���� ����
    int blocksize = DIM;

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // ȣ��Ʈ �޸� �Ҵ�
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    int gpu_sum = 0;

    // ����̽� �޸� �Ҵ�
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    int cpu_sum = recursiveReduce (tmp, size);
    printf("cpu reduce          : %d\n", cpu_sum);


    CHECK(hipMemcpy(d_idata, h_idata, bytes,                hipMemcpyHostToDevice));
    reduceNeighboredGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceNeighboredGmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);

    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceNeighboredSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceNeighboredSmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceGmem          : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceSmem          : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemDyn<<<grid.x, block, blocksize*sizeof(int)>>>(d_idata, d_odata,
            size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceSmemDyn       : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceGmemUnroll<<<grid.x / 4, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceGmemUnroll4   : %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemUnroll<<<grid.x / 4, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceSmemUnroll4   : %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemUnrollDyn<<<grid.x / 4, block, DIM*sizeof(int)>>>(d_idata,
            d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceSmemDynUnroll4: %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // �޸� ����
    free(h_idata);
    free(h_odata);
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    CHECK(hipDeviceReset());

    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}

__device__ void clock_block(clock_t *d_o, clock_t clock_count)
{
	unsigned int start_clock = (unsigned int)clock();

	clock_t clock_offset = 0;

	while (clock_offset < clock_count)
	{
		unsigned int end_clock = (unsigned int)clock();

		// The code below should work like
		// this (thanks to modular arithmetics):
		//
		// clock_offset = (clock_t) (end_clock > start_clock ?
		//                           end_clock - start_clock :
		//                           end_clock + (0xffffffffu - start_clock));
		//
		// Indeed, let m = 2^32 then
		// end - start = end + m - start (mod m).

		clock_offset = (clock_t)(end_clock - start_clock);
	}

	d_o[0] = clock_offset;
}


////////////////////////////////////////////////////////////////////////////////
// clock_block()�� ȣ���ϴ� Ŀ��.
//�� Ŀ���� ������ ��Ʈ�� �󿡼� �����ϵ��� �Ѵ�.

__global__ void kernel_A(clock_t *d_o, clock_t clock_count)
{
	clock_block(d_o, clock_count);
}
__global__ void kernel_B(clock_t *d_o, clock_t clock_count)
{
	clock_block(d_o, clock_count);
}
__global__ void kernel_C(clock_t *d_o, clock_t clock_count)
{
	clock_block(d_o, clock_count);
}
__global__ void kernel_D(clock_t *d_o, clock_t clock_count)
{
	clock_block(d_o, clock_count);
}
int simpleHyperQ(int argc, char **argv)
{
	int nstreams = 8;       // ��Ʈ�� ����
	float kernel_time = 10; // Ŀ���� ����� ms ���� �ð�
	float elapsed_time;
	int cuda_device = 0;

	char * iname = "CUDA_DEVICE_MAX_CONNECTIONS";
	setenv(iname, "4", 1); // 4 or 32
	char *ivalue = getenv(iname);
	printf("%s = %s\n", iname, ivalue);

	hipDeviceProp_t deviceProp;
	CHECK(hipGetDevice(&cuda_device));
	CHECK(hipGetDeviceProperties(&deviceProp, cuda_device));

	printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
		deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	// ȣ��Ʈ �޸� �Ҵ�
	clock_t *a = 0;
	CHECK(hipHostMalloc((void **)&a, sizeof(clock_t)));

	// ����̽� �޸� �Ҵ�
	clock_t *d_a = 0;
	CHECK(hipMalloc((void **)&d_a, 2 * nstreams * sizeof(clock_t)));

	// ��Ʈ�� ��ü�� ���� �޸� �Ҵ� �� ����
	hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

	for (int i = 0; i < nstreams; i++)
	{
		CHECK(hipStreamCreate(&(streams[i])));
	}

	// �̺�Ʈ �ڵ鷯 ����
	hipEvent_t start_event, stop_event;
	CHECK(hipEventCreate(&start_event));
	CHECK(hipEventCreate(&stop_event));

	// Target time per kernel = kernel_time ms, clockRate = in KHz
	// Target number of clocks = target time * clock frequency
#if defined(__arm__) || defined(__aarch64__)
	clock_t time_clocks = (clock_t)(kernel_time * (deviceProp.clockRate / 1000));
#else
	clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#endif
	clock_t total_clocks = 0;

	CHECK(hipEventRecord(start_event, 0));

	for (int i = 0; i < nstreams; ++i)
	{
		kernel_A <<<1, 1, 0, streams[i] >>>(&d_a[2 * i], time_clocks);
		total_clocks += time_clocks;
		kernel_B <<<1, 1, 0, streams[i] >>>(&d_a[2 * i + 1], time_clocks);
		total_clocks += time_clocks;
		kernel_C <<<1, 1, 0, streams[i] >>>(&d_a[2 * i], time_clocks);
		total_clocks += time_clocks;
		kernel_D <<<1, 1, 0, streams[i] >>>(&d_a[2 * i + 1], time_clocks);
		total_clocks += time_clocks;
	}

	// ��Ʈ�� 0 ���� �ߴ� �̺�Ʈ
	CHECK(hipEventRecord(stop_event, 0));

	// ���⼭ CPU�� GPU�� ���������� ���� �۾� ���� ����.
	// ���⼭�� ��� �۾��� �Ϸ�� ������ ����Ѵ�.

	CHECK(hipEventSynchronize(stop_event));
	CHECK(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

	printf("Expected time for serial execution of %d sets of kernels is between approx. %.3fs and %.3fs\n", nstreams, (nstreams + 1) * kernel_time / 1000.0f, 2 * nstreams *kernel_time / 1000.0f);
	printf("Expected time for fully concurrent execution of %d sets of kernels is approx. %.3fs\n", nstreams, 2 * kernel_time / 1000.0f);
	printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

	bool bTestResult = (a[0] >= total_clocks);

	for (int i = 0; i < nstreams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	free(streams);
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	hipHostFree(a);
	hipFree(d_a);

	return (bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

/////////////////////////////////////////////////////////////////////////////////
#define LOOP_COUNT 3000000

void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void *data)
{
	printf("callback from stream %d\n", *((int *)data));
}

__global__ void kernel_1()
{
	double sum = 0.0;

	for (int i = 0; i < LOOP_COUNT; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_2()
{
	double sum = 0.0;

	for (int i = 0; i < LOOP_COUNT; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_3()
{
	double sum = 0.0;

	for (int i = 0; i < LOOP_COUNT; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

__global__ void kernel_4()
{
	double sum = 0.0;

	for (int i = 0; i < LOOP_COUNT; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}



int simpleCallback(int argc, char **argv)
{
	int n_streams = 8;

	if (argc > 2) n_streams = atoi(argv[2]);

	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("> %s Starting...\n", argv[0]);
	printf("> Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
		deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	// �ִ� ���� �� ����
	char * iname = "CUDA_DEVICE_MAX_CONNECTIONS";
	setenv(iname, "8", 1);
	char *ivalue = getenv(iname);
	printf("> %s = %s\n", iname, ivalue);
	printf("> with streams = %d\n", n_streams);

	// ��Ʈ�� �Ҵ� �� �ʱ�ȭ
	hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(
		hipStream_t));

	for (int i = 0; i < n_streams; i++)
	{
		CHECK(hipStreamCreate(&(streams[i])));
	}

	dim3 block(1);
	dim3 grid(1);
	hipEvent_t start_event, stop_event;
	CHECK(hipEventCreate(&start_event));
	CHECK(hipEventCreate(&stop_event));

	int stream_ids[4];

	CHECK(hipEventRecord(start_event, 0));

	for (int i = 0; i < n_streams; i++)
	{
		stream_ids[i] = i;
		kernel_1 <<<grid, block, 0, streams[i] >>>();
		kernel_2 <<<grid, block, 0, streams[i] >>>();
		kernel_3 <<<grid, block, 0, streams[i] >>>();
		kernel_4 <<<grid, block, 0, streams[i] >>>();
		CHECK(hipStreamAddCallback(streams[i], my_callback,
			(void *)(stream_ids + i), 0));
	}

	CHECK(hipEventRecord(stop_event, 0));
	CHECK(hipEventSynchronize(stop_event));

	float elapsed_time;
	CHECK(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
	printf("Measured time for parallel execution = %.3fs\n", elapsed_time);

	// ��Ʈ�� ����
	for (int i = 0; i < n_streams; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
	}

	free(streams);

	CHECK(hipDeviceReset());

	return 0;
}
int main(int argc, char* argv[]){

   int ex=0;
   ex=atoi(argv[1]);
   printf("run ex : %d\n",ex);
   switch(ex){
    case 1:{
     printf("multithread\n");//stream
     multithread();
     break;
    }
    case 2:{
     printf("coaleascing\n");
     coaleascing(argc, argv);
     break;
    }
    case 3:{
     printf("shared_memory_reverse\n");//simple smem + sync
     shared_memory_reverse();
     break;
    }
    case 4:{
     printf("reduceSum\n");
     reduceSum(argc,argv);
     break;
    }
    case 5:{
     printf("smemSquare\n");//smem + sync
     smemSquare(argc,argv);
     break;
    }
    case 6:{
     printf("simpleHyperQ\n");//hyper q
     simpleHyperQ(argc,argv);
     break;
    }
    case 7:{
     printf("simpleCallback\n");//stream
     simpleCallback(argc,argv);
     break;
    }
    case 8:{
     printf("async\n");//simple async memcpy
     async(argc,argv);
     break;
    }
    case 9:{
     printf("data_transfer_pageable_vs_pinned\n");
     data_transfer_pageable_vs_pinned();
     break;
    }
    case 10:{
     printf("overlap\n");//stream
     overlap(argc,argv);
     break;
    }
  }
  return 0;
}