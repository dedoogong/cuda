#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
 

void printMatrix(int *C, const int nx, const int ny)
{
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%3d", ic[ix]);

        }

        ic += nx;
        printf("\n");
    }

    printf("\n");
    return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
           " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
           ix, iy, idx, A[idx]);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
 
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
 
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
 
    int *h_A;
    h_A = (int *)malloc(nBytes);
 
    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);
 
    int *d_MatA;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
 
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
 
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
 
    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    CHECK(hipGetLastError()); 

    CHECK(hipFree(d_MatA));
    free(h_A); 

    CHECK(hipDeviceReset());

    return (0);
}
