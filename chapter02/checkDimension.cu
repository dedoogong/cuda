#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
 

__global__ void checkIndex(void)
{
    printf("threadIdx:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx:(%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);

    printf("blockDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
    printf("gridDim:(%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);

}

int main(int argc, char **argv)
{ 
    int nElem = 6;
 
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);
 
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
 
    checkIndex<<<grid, block>>>();
 
    CHECK(hipDeviceReset());

    return(0);
}
