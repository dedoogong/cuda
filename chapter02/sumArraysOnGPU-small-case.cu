#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h> 

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}


void initialData(float *ip, int size)
{ 
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}


int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
 
    int dev = 0;
    CHECK(hipSetDevice(dev));
 
    int nElem = 1 << 5;
    printf("Vector size %d\n", nElem);
 
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);
 
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);
 
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));
 
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
 
    dim3 block (nElem);
    dim3 grid  (1);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);
 
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
 
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
 
    checkResult(hostRef, gpuRef, nElem);
 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
 
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());
    return(0);
}
