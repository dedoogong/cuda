#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
  
int main(int argc, char **argv)
{ 
    int nElem = 1024;
 
    dim3 block (1024);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("grid.x %d block.x %d \n", grid.x, block.x);
 
    block.x = 512;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);
 
    block.x = 256;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);
 
    block.x = 128;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);
 
    CHECK(hipDeviceReset());

    return(0);
}

